/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
 * Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 */

#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

//#include <helper_cuda.h>
/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */

#define CUDA_1D_KERNEL_LOOP(i, n)                                              \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x;                          \
       i < (n);                                                                \
       i += blockDim.x * gridDim.x)


__global__ void
print_kernel(const int N)
{
  // int i = blockDim.x * blockIdx.x + threadIdx.x;
  CUDA_1D_KERNEL_LOOP(i, N) { 
    printf("Hello %d from gridDim %d, blockDim %d, blockId %d, thread %d\n", i, gridDim.x, blockDim.x, blockIdx.x, threadIdx.x);
  }
}

/**
 * Host main routine
 */
int main(void)
{
  print_kernel<<<5, 10>>>(50);

  hipDeviceSynchronize();

  return 0;
}

