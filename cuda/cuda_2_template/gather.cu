#include <stdio.h>
#include <iostream>
#include "gather.cu.h"

int main() {
  // 3 by 4
  float p_src[12];
  for (int i = 0; i < 12; ++i) p_src[i] = float(i);

  int p_index[2] = {1, 2};
  
  float p_out[8];

  // implemented by function
  // specialization at runtime
  // Gather<float>(p_src, p_index, p_out, 4, 2, true);

  Gather2<float, true> g_functor1;
  g_functor1(p_src, p_index, p_out, 4, 2);

  printf("CPU gather result\n");
  for (int i = 0; i < 8; ++i)
    printf("%f\n", p_out[i]);
  printf("\n");

  // Try on GPU
  float* d_src = NULL;
  int* d_index = NULL;
  float* d_out = NULL;
  hipMalloc((void**)&d_src, 12 * sizeof(float));
  hipMalloc((void**)&d_index, 2 * sizeof(int));
  hipMalloc((void**)&d_out, 8 * sizeof(float));

  hipMemcpy(d_src, p_src, 12 * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_index, p_index, 2 * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_out, p_out, 8 * sizeof(float), hipMemcpyHostToDevice);
  
  hipMemcpy(d_src, p_src, 12 * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(d_index, p_index, 2 * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(d_out, p_out, 8 * sizeof(float), hipMemcpyDeviceToHost);
  
  printf("Source:\n");
  for (int i = 0; i < 12; ++i)
    printf("%f\n", p_src[i]);
  printf("\n");
  printf("Index:\n");
  for (int i = 0; i < 2; ++i)
    printf("%d\n", p_index[i]);
  printf("\n");
  printf("Output:\n");
  for (int i = 0; i < 8; ++i)
    printf("%f\n", p_out[i]);
  printf("\n");

  // Gather<float>(d_src, d_index, d_out, 4, 2, false);

  Gather2<float, true> g_functor2;
  g_functor2(p_src, p_index, p_out, 4, 2);
  
  float g_src[12];
  int g_index[2];
  float g_out[8];
  hipMemcpy(g_src, d_src, 12 * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(g_index, d_index, 2 * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(g_out, d_out, 8 * sizeof(float), hipMemcpyDeviceToHost);

  printf("Source:\n");
  for (int i = 0; i < 12; ++i)
    printf("%f\n", g_src[i]);
  printf("Index:\n");
  for (int i = 0; i < 2; ++i)
    printf("%d\n", g_index[i]);
  printf("Output:\n");
  for (int i = 0; i < 8; ++i)
    printf("%f\n", g_out[i]);

  return 0;
}
